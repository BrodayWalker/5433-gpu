// gcc BrodayWalkerSequential.c -o BrodayWalkerSequential.exe


#include <hip/hip_runtime.h>
#include <stdio.h>

enum N {N = 32};

void print(int [][N], int);

int main()
{
    // Declarations
    int A[N][N], B[N][N], C[N][N];

    // Fill arrays A and C
    // Array C will be filled with 0s
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            A[i][j] = i;
            C[i][j] = 0;
        }
    
    // Fill B
    int row = N - 1;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            B[i][j] = row;
        row--;        
    }

    // Print array A
    printf("Array A: \n");
    print(A, N);

    printf("\n\nArray B: \n");
    print(B, N);

    printf("\n\nArray C: \n");
    print(C, N);
    
    return 0;
}

// function: print()
// parameters: int [][N], int width
// The print function prints a 2D array
void print(int ray[][N], int width)
{
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
            printf("%d ", ray[i][j]);
        printf("\n");
    }
}