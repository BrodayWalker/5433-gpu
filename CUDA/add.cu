
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>

using namespace std;

const int N = 4096;
const int BLOCKSIZE = 1024;

__global__
void add_me(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
 	c[i] = a[i] + b[i];
}

int main()
{
    ofstream outfile;
    outfile.open("output.txt");

	int a[N] = {0};
 	int b[N];
    int c[N] = {0};
    int sum = 0;

    // Load b with 1s
    for (int i = 0; i < N; i++)
        b[i] = 1;

 	int *ad;
 	int *bd;
    int *cd;
 	const int isize = N*sizeof(int);

 	hipMalloc( (void**)&ad, isize );
 	hipMalloc( (void**)&bd, isize );
    hipMalloc( (void**)&cd, isize );
 	hipMemcpy( ad, a, isize, hipMemcpyHostToDevice );
 	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );
    hipMemcpy( cd, c, isize, hipMemcpyHostToDevice );

 	dim3 dimGrid( 4, 1 ); 	
	dim3 dimBlock( BLOCKSIZE, 1 );

 	add_me<<<dimGrid, dimBlock>>>(ad, bd, cd);

 	hipMemcpy( c, cd, isize, hipMemcpyDeviceToHost );
 	hipFree( ad );
	hipFree( bd );
    hipFree( cd );

    for (int i = 0; i < N; i++)
        sum += c[i];

    cout << "The sum is: " << sum << '\n';
        

 	return EXIT_SUCCESS;
}
